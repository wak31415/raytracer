#include "hip/hip_runtime.h"
#include "cuda/scene_objects.cuh"
#include "cuda/projection_helpers.cuh"
#include <stdio.h>

__global__ void raytrace_spheres_kernel(Sphere* spheres, 
                                        size_t sphere_count, 
                                        int* visible, 
                                        CU_Vector3f* vertices, 
                                        CU_Vector3f* normals,
                                        CU_Vector3f* image, 
                                        CU_Matrix<3> cam_rot,
                                        CU_Vector3f camera_pos,
                                        CU_Matrix<3> K,
                                        uint width,
                                        uint height) 
{
    uint u_x = blockDim.x * blockIdx.x + threadIdx.x;
    uint u_y = blockDim.y * blockIdx.y + threadIdx.y;

    uint idx = u_y * width + u_x;

    if (u_x >= width || u_y >= height) 
        return;
    
    if (idx == 0) {
        printf("fx, fy = (%.3f, %.3f)\n", K[0*3+0], K[1*3+1]);
        printf("Color sphere 1: (%.f, %.f, %.f)\n", spheres[1].color[0], spheres[1].color[1], spheres[1].color[2]);
        printf("Camera position: (%.3f, %.3f, %.3f)\n", camera_pos[0], camera_pos[1], camera_pos[2]);
        // printf("Camera extrinsics:\n");
        // printf("fx, fy = (%.3f, %.3f)\n", camera->K[0*3+0], camera->K[1*3+1]);
    }



    // obtain ray direction
    CU_Vector3f ray_dir = pixel_to_camera(u_x, u_y, 1.f, K);
    ray_dir.normalize();
    ray_dir = cam_rot*ray_dir;

    if(idx < 10) {
        printf("Ray direction: (%.3f, %.3f, %.3f)\n", ray_dir[0], ray_dir[1], ray_dir[2]);
    }

    float min_dist = 0.f;
    int min_id = -1;

    for(size_t i = 0; i < sphere_count; i++) {
        CU_Vector3f sphere_to_cam = camera_pos - spheres[i].pos;
        float dot_prod = dot(ray_dir, sphere_to_cam);
        
        float delta = powf(dot_prod, 2.f) - (powf(sphere_to_cam.norm(), 2.f) - powf(spheres[i].radius, 2.f));

        if (delta >= 0) {
            float t;
            float t1 = - dot_prod - sqrtf(delta);
            float t2 = - dot_prod + sqrtf(delta);
            
            t = t1 >= 0 ? t1 : t2;

            if (min_id == -1 || t < min_dist) {
                min_dist = t;
                min_id = i;
            }
        }
    }

    // if(idx < 10) {
    //     printf("Intersecting with sphere %d\n", min_id);
    // }

    visible[idx] = min_id;

    if(min_id >= 0) {
        CU_Vector3f P = camera_pos + min_dist*ray_dir;
        vertices[idx] = P;
        CU_Vector3f tmp = P - spheres[min_id].pos;
        normals[idx] = (1/tmp.norm()) * tmp;
        image[idx] = spheres[min_id].color;
    }
}

void raytrace_spheres(Sphere* spheres, size_t sphere_count, int* visible, CU_Vector3f* vertices, CU_Vector3f* normals, CU_Vector3f* image, Camera* camera) {
    size_t vertex_count = camera->width * camera->height;

    Sphere* d_spheres;
    CU_Matrix<4> d_cam_rot;
    CU_Vector3f d_cam_trans;
    CU_Matrix<3> d_K;
    int* d_visible;
    CU_Vector3f* d_image;
    CU_Vector3f* d_vertices;
    CU_Vector3f* d_normals;

    CU_Matrix<3> cam_rot = camera->E.get_rotation();
    CU_Vector3f cam_trans = camera->E.get_translation();

    hipMalloc((void**)&d_spheres, sphere_count*sizeof(struct Sphere));
    // hipMalloc((void**)&d_cam_rot, sizeof(CU_Matrix<3>));
    // hipMalloc((void**)&d_cam_trans, sizeof(CU_Vector3f));
    // hipMalloc((void**)&d_K, sizeof(CU_Matrix<3>));
    hipMalloc((void**)&d_visible, vertex_count*sizeof(int));
    hipMalloc((void**)&d_image, vertex_count*sizeof(CU_Vector3f));
    hipMalloc((void**)&d_vertices, vertex_count*sizeof(CU_Vector3f));
    hipMalloc((void**)&d_normals, vertex_count*sizeof(CU_Vector3f));

    hipMemcpy(d_spheres, spheres, sphere_count*sizeof(struct Sphere), hipMemcpyHostToDevice);
    // hipMemcpy(d_cam_rot, cam_rot, sizeof(CU_Matrix<3>), hipMemcpyHostToDevice);
    // hipMemcpy(d_cam_trans, cam_trans, sizeof(CU_Vector3f), hipMemcpyHostToDevice);
    // hipMemcpy(d_K, camera-K, sizeof(CU_Matrix<3>), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32,32);
    dim3 blocksPerGrid((camera->width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (camera->height + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    raytrace_spheres_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_spheres,
        sphere_count,
        d_visible,
        d_vertices,
        d_normals,
        d_image,
        cam_rot,
        cam_trans,
        camera->K,
        512,
        512
    );
    hipDeviceSynchronize();

    hipMemcpy(visible, d_visible, vertex_count*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(image, d_image, vertex_count*sizeof(CU_Vector3f), hipMemcpyDeviceToHost);
    hipMemcpy(vertices, d_vertices, vertex_count*sizeof(CU_Vector3f), hipMemcpyDeviceToHost);
    hipMemcpy(normals, d_normals, vertex_count*sizeof(CU_Vector3f), hipMemcpyDeviceToHost);

    hipFree(d_spheres);
    // hipFree(d_cam_rot);
    // hipFree(d_cam_trans);
    // hipFree(d_K);
    hipFree(d_visible);
    hipFree(d_image);
    hipFree(d_vertices);
    hipFree(d_normals);
}