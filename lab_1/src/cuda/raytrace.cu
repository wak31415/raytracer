#include "hip/hip_runtime.h"
#include "cuda/scene_objects.cuh"
#include "cuda/projection_helpers.cuh"
#include <stdio.h>

#define PI 3.14159265358979
#define GAMMA 2.2

__device__ CU_Vector3f clamp_color(CU_Vector3f color) {
    CU_Vector3f res;
    res[0] = fminf(color[0], 1.f);
    res[1] = fminf(color[1], 1.f);
    res[2] = fminf(color[2], 1.f);
    return res;
}

__device__ CU_Vector3f gamma_correct(CU_Vector3f color) {
    CU_Vector3f res;
    float exponent = 1.f/GAMMA;
    res[0] = powf(color[0], exponent);
    res[1] = powf(color[1], exponent);
    res[2] = powf(color[2], exponent);
    return res;
}

__device__ CU_Vector3f get_intersection(Sphere* spheres, 
                                        size_t sphere_count, 
                                        CU_Vector3f ray, 
                                        CU_Vector3f start, 
                                        int* intersect_id) 
{
    float min_dist = 0.f;

    for(size_t i = 0; i < sphere_count; i++) {
        CU_Vector3f O_C = start - spheres[i].pos;

        float ray_dot_O_C = dot(ray, O_C);
        float delta = powf(ray_dot_O_C, 2.f) - powf(O_C.norm(), 2.f) + powf(spheres[i].radius, 2.f);

        if (delta >= 0) {
            float t;
            float t1 = - ray_dot_O_C - sqrtf(delta);
            float t2 = - ray_dot_O_C + sqrtf(delta);

            if (t2 >= 0) {
                t = t1 >= 0 ? t1 : t2;

                if (*intersect_id < 0 || t < min_dist) {
                    min_dist = t;
                    *intersect_id = i;
                }
            }
        }
    }
    return start + min_dist*ray;
}

__device__ bool is_visible(Sphere* spheres, size_t sphere_count, CU_Vector3f origin, CU_Vector3f target) {
    int intersect_id = -1;
    CU_Vector3f ray = target - origin;
    ray.normalize();
    float t = get_intersection(spheres, sphere_count, ray, origin, &intersect_id);
    if (t < (target - origin).norm() && intersect_id >= 0) return false;
    return true;
}

__device__ CU_Vector3f reflected_direction(CU_Vector3f ray, CU_Vector3f normal) {
    return ray - 2 * dot(ray, normal) * normal;
}

__device__ CU_Vector3f get_color(CU_Vector3f P, CU_Vector3f ray, int ray_depth) {
    if (ray_depth < 0) return CU_Vector3f();


}

__global__ void raytrace_spheres_kernel(Sphere* spheres, 
                                        size_t sphere_count, 
                                        Light* lights,
                                        size_t light_count,
                                        int* visible, 
                                        CU_Vector3f* vertices, 
                                        CU_Vector3f* normals,
                                        CU_Vector3f* image, 
                                        CU_Matrix<3> cam_rot,
                                        CU_Vector3f camera_pos,
                                        CU_Matrix<3> K,
                                        uint width,
                                        uint height) 
{
    uint u_x = blockDim.x * blockIdx.x + threadIdx.x;
    uint u_y = blockDim.y * blockIdx.y + threadIdx.y;

    uint idx = u_y * width + u_x;

    if (u_x >= width || u_y >= height) 
        return;

    // obtain ray direction
    CU_Vector3f ray_dir = pixel_to_camera(u_x, u_y, 1.f, K);
    ray_dir.normalize();
    ray_dir = cam_rot*ray_dir;

    int min_id = -1;
    CU_Vector3f P = get_intersection(spheres, sphere_count, ray_dir, camera_pos, &min_id);

    visible[idx] = min_id;

    if(min_id >= 0) {
        vertices[idx] = P;
        CU_Vector3f tmp = P - spheres[min_id].pos;
        normals[idx] = (1/tmp.norm()) * tmp;

        // Normalized vector point --> light
        CU_Vector3f S_P = lights[0].pos - P;
        float d = S_P.norm();
        CU_Vector3f w_i = 1.f/d * S_P;

        float N_wi_dot = max(dot(normals[idx], w_i), 0.f);

        // check if the light is visible from P
        bool P_visible = is_visible(spheres, sphere_count, P+0.001*normals[idx], lights[0].pos);

        CU_Vector3f L = clamp_color(lights[0].I / (4*powf(PI*d, 2.f)) * spheres[min_id].color * P_visible * N_wi_dot);

        image[idx] = gamma_correct(L);
    }
}

void raytrace_spheres(Sphere* spheres, size_t sphere_count, Light* lights, size_t light_count, int* visible, CU_Vector3f* vertices, CU_Vector3f* normals, CU_Vector3f* image, Camera* camera) {
    size_t vertex_count = camera->width * camera->height;

    Sphere* d_spheres;
    Light* d_lights;
    CU_Matrix<4> d_cam_rot;
    CU_Vector3f d_cam_trans;
    CU_Matrix<3> d_K;
    int* d_visible;
    CU_Vector3f* d_image;
    CU_Vector3f* d_vertices;
    CU_Vector3f* d_normals;

    CU_Matrix<3> cam_rot = camera->E.get_rotation();
    CU_Vector3f cam_trans = camera->E.get_translation();

    hipMalloc((void**)&d_spheres, sphere_count*sizeof(struct Sphere));
    hipMalloc((void**)&d_lights, light_count*sizeof(struct Light));
    // hipMalloc((void**)&d_cam_rot, sizeof(CU_Matrix<3>));
    // hipMalloc((void**)&d_cam_trans, sizeof(CU_Vector3f));
    // hipMalloc((void**)&d_K, sizeof(CU_Matrix<3>));
    hipMalloc((void**)&d_visible, vertex_count*sizeof(int));
    hipMalloc((void**)&d_image, vertex_count*sizeof(CU_Vector3f));
    hipMalloc((void**)&d_vertices, vertex_count*sizeof(CU_Vector3f));
    hipMalloc((void**)&d_normals, vertex_count*sizeof(CU_Vector3f));

    hipMemcpy(d_spheres, spheres, sphere_count*sizeof(struct Sphere), hipMemcpyHostToDevice);
    hipMemcpy(d_lights, lights, light_count*sizeof(struct Light), hipMemcpyHostToDevice);
    // hipMemcpy(d_cam_rot, cam_rot, sizeof(CU_Matrix<3>), hipMemcpyHostToDevice);
    // hipMemcpy(d_cam_trans, cam_trans, sizeof(CU_Vector3f), hipMemcpyHostToDevice);
    // hipMemcpy(d_K, camera-K, sizeof(CU_Matrix<3>), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32,32);
    dim3 blocksPerGrid((camera->width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (camera->height + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    raytrace_spheres_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_spheres,
        sphere_count,
        d_lights,
        light_count,
        d_visible,
        d_vertices,
        d_normals,
        d_image,
        cam_rot,
        cam_trans,
        camera->K,
        512,
        512
    );
    hipDeviceSynchronize();

    hipMemcpy(visible, d_visible, vertex_count*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(image, d_image, vertex_count*sizeof(CU_Vector3f), hipMemcpyDeviceToHost);
    hipMemcpy(vertices, d_vertices, vertex_count*sizeof(CU_Vector3f), hipMemcpyDeviceToHost);
    hipMemcpy(normals, d_normals, vertex_count*sizeof(CU_Vector3f), hipMemcpyDeviceToHost);

    hipFree(d_spheres);
    hipFree(d_lights);
    // hipFree(d_cam_rot);
    // hipFree(d_cam_trans);
    // hipFree(d_K);
    hipFree(d_visible);
    hipFree(d_image);
    hipFree(d_vertices);
    hipFree(d_normals);
}