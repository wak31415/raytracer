#include "hip/hip_runtime.h"
#include "scene_objects.cuh"
#include "projection_helpers.cuh"
#include <iostream>
#include <string>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <ctime>
#include <hip/hip_math_constants.h>

#define GAMMA 2.2
#define MAX_RAY_DEPTH 7

#define INDIRECT_LIGHTING 1

__global__ void initialize_states(unsigned int seed, size_t width, hiprandState_t* states) {
    uint u_x = blockDim.x * blockIdx.x + threadIdx.x;
    uint u_y = blockDim.y * blockIdx.y + threadIdx.y;

    uint idx = u_y * width + u_x;

    /* we have to initialize the state */
    hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
                idx, /* the sequence number should be different for each core (unless you want all
                               cores to get the same sequence of numbers for some reason - use thread id! */
                0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
                &states[idx]);
}

/* Get two independent normally distributed samples, store in x and y */
__device__ void boxMueller(hiprandState_t* states, unsigned int idx, float std, float* x, float* y) {
    float r1 = hiprand_uniform(&states[idx]);
    float r2 = hiprand_uniform(&states[idx]);

    *x = sqrt(-2 * log(r1)) * cos(2 * M_PI * r2) * std;
    *y = sqrt(-2 * log(r1)) * sin(2 * M_PI * r2) * std;
}

__device__ CU_Vector3f gamma_correct(CU_Vector3f color) {
    CU_Vector3f res;
    float exponent = 1.f/GAMMA;
    res[0] = powf(color[0], exponent);
    res[1] = powf(color[1], exponent);
    res[2] = powf(color[2], exponent);
    return res;
}

__device__ float get_distance(Sphere* spheres, 
                              size_t sphere_count,
                              CU_Vector3f ray,
                              CU_Vector3f start,
                              int* intersect_id)
{
    float min_dist = HIP_INF_F;

    for(size_t i = 0; i < sphere_count; i++) {
        CU_Vector3f O_C = start - spheres[i].pos;

        float ray_dot_O_C = dot(ray, O_C);
        float O_C_norm = O_C.norm();
        float R = spheres[i].radius;
        float delta = ray_dot_O_C*ray_dot_O_C - O_C_norm*O_C_norm + R*R;

        if (delta >= 0) {
            float t;
            float t1 = - ray_dot_O_C - sqrtf(delta);
            float t2 = - ray_dot_O_C + sqrtf(delta);

            if (t2 >= 0) {
                t = t1 >= 0 ? t1 : t2;

                if (*intersect_id < 0 || t < min_dist) {
                    min_dist = t;
                    *intersect_id = i;
                }
            }
        }
    }
    return min_dist;
}

__device__ float get_distance(Triangle* triangles, 
                              size_t triangle_count,
                              CU_Vector3f ray,
                              CU_Vector3f start, 
                              int* intersect_id)
{
    float min_dist = HIP_INF_F;

    for(size_t i = 0; i < triangle_count; i++) {
        Triangle T = triangles[i];
        CU_Vector3f e1 = T.B - T.A;
        CU_Vector3f e2 = T.C - T.A;
        CU_Vector3f A_O_cross_u = (T.A - start).cross(ray);
        CU_Vector3f N = e1.cross(e2);
        float rayDotN = dot(ray, N);

        float beta = dot(e2, A_O_cross_u) / rayDotN;
        float gamma = - dot(e1, A_O_cross_u) / rayDotN;

        float alpha = 1.f - beta - gamma;
        if (alpha >= 0 && beta >= 0 && gamma >= 0) {
            float t = dot(T.A - start, T.N) / rayDotN;
            if (*intersect_id < 0 || t < min_dist) {
                min_dist = t;
                *intersect_id = i;
            }
        }
    }
    return min_dist;
}

__device__ CU_Vector3f get_intersection(Sphere* spheres, 
                                        size_t sphere_count,
                                        Triangle* triangles,
                                        size_t triangle_count,
                                        CU_Vector3f ray,
                                        CU_Vector3f start, 
                                        int* sphere_id,
                                        int* triangle_id) 
{
    float min_dist = HIP_INF_F;
    float min_dist_spheres = get_distance(spheres, sphere_count, ray, start, sphere_id);
    float min_dist_triangles = get_distance(triangles, triangle_count, ray, start, triangle_id);
    if(min_dist_spheres < min_dist_triangles) {
        min_dist = min_dist_spheres;
        *triangle_id = -1;
    } else {
        min_dist = min_dist_triangles;
        *sphere_id = -1;
    }
    if (__isinff(min_dist)) return CU_Vector3f(HIP_INF_F, HIP_INF_F, HIP_INF_F);
    return start + min_dist*ray;
}

__device__ bool is_visible(Sphere* spheres, size_t sphere_count, Triangle* triangles, size_t triangle_count, CU_Vector3f origin, CU_Vector3f target) {
    int sphere_id = -1;
    int triangle_id = -1;
    CU_Vector3f ray = target - origin;
    ray.normalize();
    CU_Vector3f P = get_intersection(spheres, sphere_count, triangles, triangle_count, ray, origin, &sphere_id, &triangle_id);
    if ((P - origin).norm() < (target - origin).norm()) return false;
    return true;
}

__device__ CU_Vector3f reflected_direction(CU_Vector3f ray, CU_Vector3f normal) {
    return ray - 2 * dot(ray, normal) * normal;
}

/**
 * Generates a random ray for indirect lighting
 **/
__device__ CU_Vector3f random_cos(hiprandState_t* states, CU_Vector3f normal, unsigned int idx) {
    float r1 = hiprand_uniform(&states[idx]);
    float r2 = hiprand_uniform(&states[idx]);

    float x = cosf(2*M_PI*r1)*sqrtf(1-r2);
    float y = sinf(2*M_PI*r1)*sqrtf(1-r2);
    float z = sqrtf(r2);

    // generate orthogonal vectors T1 and T2
    int k = normal.argmin_abs(); 
    int i = (int)fmod(k + 1.f, 3.f);
    int j = (int)fmod(k + 2.f, 3.f);

    // T1
    CU_Vector3f T1;
    T1[i] = normal[j];
    T1[j] = -normal[i];
    T1[k] = 0.f;
    T1.normalize();

    // T2
    CU_Vector3f T2 = normal.cross(T1);
    T2.normalize();

    return x*T1 + y*T2 + z*normal;
}

__device__ CU_Vector3f get_color(Sphere* spheres, 
                                 size_t sphere_count,
                                 Triangle* triangles,
                                 size_t triangle_count, 
                                 Light* lights,
                                 size_t light_count,
                                 CU_Vector3f start, 
                                 CU_Vector3f ray,
                                 bool* terminate_early,
                                 hiprandState_t* states,
                                 unsigned int idx)
{
    int depth = 0;
    CU_Vector3f L;
    CU_Vector3f albedo(1.f, 1.f, 1.f);

    while(depth < MAX_RAY_DEPTH) {
        int sphere_id = -1;
        int triangle_id = -1;
        CU_Vector3f P = get_intersection(spheres, sphere_count, triangles, triangle_count, ray, start, &sphere_id, &triangle_id);
        
        if(sphere_id >= 0 || triangle_id >= 0) {
            Material material;
            CU_Vector3f N;
            if(sphere_id >= 0) {
                material = spheres[sphere_id].material;
                N = P - spheres[sphere_id].pos;
                N.normalize();
            } else {
                material = triangles[triangle_id].material;
                N = triangles[triangle_id].N;
            }

            // Diffuse
            if(material.type == DIFFUSE) {
                // First surface we reach is diffuse, single ray is sufficient
                if(!INDIRECT_LIGHTING) {
                    if(depth == 0) *terminate_early = true;
                }

                // Normalized vector point --> light
                CU_Vector3f S_P = lights[0].pos - P;
                float d = S_P.norm();
                CU_Vector3f w_i = 1.f/d * S_P;

                float N_wi_dot = max(dot(N, w_i), 0.f);

                // check if the light is visible from P
                bool P_visible = is_visible(spheres, sphere_count, triangles, triangle_count, P+0.01*N, lights[0].pos);

                CU_Vector3f direct = lights[0].I / (4*M_PI*M_PI*d*d) * material.color * P_visible * N_wi_dot;
                
                L += albedo * direct;

                if(!INDIRECT_LIGHTING) return L;

                albedo *= material.color;

                start = P + 0.01*N;
                ray = random_cos(states, N, idx);
            }

            // Mirror
            else if(material.type == MIRROR) {
                CU_Vector3f reflected_ray = reflected_direction(ray, N);
                start = P + 0.01*N;
                ray = reflected_ray;
            }

            // Glass 
            else if(material.type == GLASS) {
                float ro = material.ro;
                float ri = material.ri;

                float wi_N_dot = dot(ray, N);

                // Fresnel
                float k0 = (ro - ri)*(ro - ri) / ((ro + ri)*(ro + ri));
                float R = k0 + (1.f-k0)*powf(1-abs(wi_N_dot), 5.f);

                // generate random numer
                float r = hiprand_uniform(&states[idx]);

                if(r < R) {
                    CU_Vector3f reflected_ray = reflected_direction(ray, N);
                    start = P+0.01*N;
                    ray = reflected_ray;

                } else {
                    CU_Vector3f wt_T;
                    CU_Vector3f wt_N;

                    // ray coming from the inside
                    if(wi_N_dot > 0) {
                        float tmp1 = ro;
                        ro = ri;
                        ri = tmp1;
                        N = -1.f * N;
                        wi_N_dot = dot(ray, N);
                    }

                    float tmp = 1.f - (ro/ri)*(ro/ri)*(1.f - wi_N_dot*wi_N_dot);

                    if(tmp < 0) {
                        // Total internal reflection
                        CU_Vector3f reflected_ray = reflected_direction(ray, N);
                        
                        start = P + 0.01*N;
                        ray = reflected_ray;
                    } else {
                        wt_T = ro / ri * (ray - wi_N_dot*N);
                        wt_N = - sqrtf(tmp)*N;
                        CU_Vector3f wt = wt_T + wt_N;

                        start = P - 0.01*N;
                        ray = wt;
                        // n1 = n2;
                    }
                }
            }
        }
        else {
            return L;
        }
        depth = depth + 1;
    }
    return L;
}

__global__ void raytrace_spheres_kernel(Sphere* spheres, 
                                        size_t sphere_count, 
                                        Triangle* triangles,
                                        size_t triangle_count,
                                        Light* lights,
                                        size_t light_count,
                                        CU_Vector3f* image, 
                                        CU_Matrix<3> cam_rot,
                                        CU_Vector3f camera_pos,
                                        CU_Matrix<3> K,
                                        uint width,
                                        uint height,
                                        uint num_rays,
                                        hiprandState_t* states,
                                        volatile int* progress) 
{
    uint u_x = blockDim.x * blockIdx.x + threadIdx.x;
    uint u_y = blockDim.y * blockIdx.y + threadIdx.y;

    uint idx = u_y * width + u_x;

    if (u_x >= width || u_y >= height) 
        return;

    bool terminate_early = false;
    CU_Vector3f color(0.f, 0.f, 0.f);

    for(int i = 0; i < num_rays; i++) {
        float dx=1.f, dy=1.f;

        // randomizing ray direction for anti-aliasing
        while(abs(dx) > 0.5f || abs(dy) > 0.5f)
            boxMueller(states, idx, 1.f, &dx, &dy);

        if(abs(dx) <= 0.5f && abs(dy) <= 0.5f) {
            // obtain ray direction
            // if(idx==0) printf("Ray %d\n", i);
            CU_Vector3f ray_dir = pixel_to_camera(u_x+0.5f+dx, u_y+0.5f+dx, 1.f, K);
            ray_dir.normalize();
            ray_dir = cam_rot*ray_dir;

            color += get_color(spheres, sphere_count, triangles, triangle_count, lights, light_count, camera_pos, ray_dir, &terminate_early, states, idx);
        }
    }
    image[idx] = gamma_correct((1.f/num_rays) * color);

    // Update progress
    if (!(threadIdx.x || threadIdx.y)){
        atomicAdd((int *)progress, 1);
        __threadfence_system();
    }
}

void raytrace_spheres(Sphere* spheres, 
                      size_t sphere_count, 
                      Triangle* triangles, 
                      size_t triangle_count, 
                      Light* lights, 
                      size_t light_count, 
                      CU_Vector3f* image, 
                      Camera* camera) 
{
    size_t vertex_count = camera->width * camera->height;

    hiprandState_t* d_states;
    Sphere* d_spheres;
    Triangle* d_triangles;
    Light* d_lights;
    CU_Matrix<4> d_cam_rot;
    CU_Vector3f d_cam_trans;
    CU_Matrix<3> d_K;
    CU_Vector3f* d_image;

    CU_Matrix<3> cam_rot = camera->E.get_rotation();
    CU_Vector3f cam_trans = camera->E.get_translation();

    hipMalloc((void**)&d_states, vertex_count*sizeof(hiprandState_t));
    hipMalloc((void**)&d_spheres, sphere_count*sizeof(Sphere));
    hipMalloc((void**)&d_triangles, triangle_count*sizeof(Triangle));
    hipMalloc((void**)&d_lights, light_count*sizeof(struct Light));
    hipMalloc((void**)&d_image, vertex_count*sizeof(CU_Vector3f));

    hipMemcpy(d_spheres, spheres, sphere_count*sizeof(Sphere), hipMemcpyHostToDevice);
    hipMemcpy(d_triangles, triangles, triangle_count*sizeof(Triangle), hipMemcpyHostToDevice);
    hipMemcpy(d_lights, lights, light_count*sizeof(struct Light), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(8, 16);
    dim3 blocksPerGrid((camera->width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (camera->height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    volatile int *d_data, *h_data;
    hipSetDeviceFlags(hipDeviceMapHost);
    hipHostAlloc((void **)&h_data, sizeof(int), hipHostMallocMapped);
    hipHostGetDevicePointer((int **)&d_data, (int *)h_data, 0);
    *h_data = 0;

    hipEvent_t start, stop;
    hipEventCreate(&start); hipEventCreate(&stop);

    hipEventRecord(start);

    clock_t t = std::clock();

    initialize_states<<<blocksPerGrid, threadsPerBlock>>>(time(0), camera->width, d_states);
    
    hipDeviceSynchronize();

    raytrace_spheres_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_spheres,
        sphere_count,
        d_triangles,
        triangle_count,
        d_lights,
        light_count,
        d_image,
        cam_rot,
        cam_trans,
        camera->K,
        camera->width,
        camera->height,
        camera->num_rays,
        d_states,
        d_data
    );
    hipEventRecord(stop);

    unsigned int num_blocks = blocksPerGrid.x*blocksPerGrid.y;
    float my_progress = 0.0f;
    do{
        hipEventQuery(stop);  // may help WDDM scenario
        int value1 = *h_data;
        float kern_progress = (float)value1/(float)num_blocks;
        if ((kern_progress - my_progress)> 0.02f) {
            float time_passed = (float)(std::clock() - t)/static_cast<float>(CLOCKS_PER_SEC);
            float eta = time_passed / kern_progress - time_passed;
            std::cout << "\rProgress: [";
            for(size_t i = 0; i < 20; i++) {
                if(i/20.f < my_progress) {
                    std::cout << "#";
                } else {
                    std::cout << " ";
                }
            }
            char s[10];
            memset((void*)s, 0, 10*sizeof(char));
            std::snprintf(s, 10*sizeof(char), "%.2f", eta);
            std::cout << "] - "<< static_cast<int>(kern_progress*100) << "% \t Remaining: " << (char*)s << "s" << std::flush;
            // fflush(stdout);
            my_progress = kern_progress;
        }
    }
    while (my_progress < 0.98f);
    printf("\n");

    hipEventSynchronize(stop);
    float et;
    hipEventElapsedTime(&et, start, stop);
    hipDeviceSynchronize();
    printf("Finished raytracing in %.3f seconds.\n", (double)(std::clock() - t)/CLOCKS_PER_SEC);//et/1000.f);

    hipMemcpy(image, d_image, vertex_count*sizeof(CU_Vector3f), hipMemcpyDeviceToHost);

    hipFree(d_spheres);
    hipFree(d_triangles);
    hipFree(d_lights);
    hipFree(d_image);
}